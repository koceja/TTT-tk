#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"
#include "kittens.cuh"
#include <torch/torch.h>
#include <hip/hip_math_constants.h>
#include <c10/util/BFloat16.h>

#define ASSERT(cond) if (!(cond)) { printf("Assertion failed: %s\n", #cond); return 1; }

using namespace kittens;

constexpr int NUM_WORKERS = 4; // TODO: why is one warpgroup optimal?

template<int B=1, int H=1, int N=16, int F=64, int K=4, int TP=4>
__launch_bounds__(NUM_WORKERS*WARP_THREADS, 1)
__cluster_dims__(TP)
__global__ void ttt_tp_forward_ker(
    // TODO: make B and H runtime dimensions by instantiating templates with -1 args
    const __grid_constant__ gl<bf16, B, H, N, F> XQ_gl,
    const __grid_constant__ gl<bf16, B, H, N, F> XK_gl,
    const __grid_constant__ gl<bf16, B, H, N, F> XV_gl,
    const __grid_constant__ gl<bf16, B, H, F, F*K> W1_gl,
    const __grid_constant__ gl<bf16, B, H, F*K, F> W2_gl,
    const __grid_constant__ gl<bf16, B, H, N, F> out_gl,
    bool *signal
) {
    int b = blockIdx.y;
    int h = blockIdx.z;
    cooperative_groups::cluster_group cluster = cooperative_groups::this_cluster();
    int tp = cluster.block_rank();
    
    rt_bf<N, F, ducks::rt_layout::row> XK;
    rt_bf<F, F*K/TP, ducks::rt_layout::col> W1;
    rt_fl<N, F*K/TP, ducks::rt_layout::row> Z1;
    rt_bf<N, F*K/TP, ducks::rt_layout::row> X2;
    rt_bf<F*K/TP, F, ducks::rt_layout::col> W2;
    rt_fl<N, F, ducks::rt_layout::row> out;
    st_fl<N, F> out_part;
    
    load(XK, XK_gl, {b, h, 0, 0});
    load(W1, W1_gl, {b, h, 0, tp*(F*K/TP)});
    zero(Z1);
    mma_AB(Z1, XK, W1, Z1);
    copy(X2, Z1); //dtype conversion
    mma_AB(out, X2, W2, out);
    
    // store(out_part, out);
    for (int i = 0; i < N*F; i+=blockDim.x)
        out_part[i + threadIdx.x] = tp;

    extern __shared__ KITTENS_DEFAULT_ALIGN float shm[N * F];
    float *dsmem = cluster.map_shared_rank(&shm[0], 0);
        for (int i = 0; i < N*F; i+=blockDim.x)
            atomicAdd(&dsmem[i + threadIdx.x], out_part[i + threadIdx.x]);
    cluster.sync();

    if (tp == 0 && threadIdx.x == 0)
        for (int i = 0; i < N*F; i++)
            printf("%.1f ", shm[i]);
    cluster.sync();

    *signal = true;
}

extern torch::Tensor ttt_tp_forward(
    const torch::Tensor XQ,
    const torch::Tensor XK,
    const torch::Tensor XV,
    const torch::Tensor W1,
    const torch::Tensor W2,
    const torch::Tensor out
) {
    constexpr int B = 1, H = 1, N = 16, F = 64, K = 4, TP = 4;

    // TODO: better macro
    TORCH_CHECK(XQ.device().is_cuda() && XQ.is_contiguous() && XQ.dim() == 4 && XQ.size(0) == B && XQ.size(1) == H && XQ.size(2) == N && XQ.size(3) == F, "XQ");
    TORCH_CHECK(XK.device().is_cuda() && XK.is_contiguous() && XK.dim() == 4 && XK.size(0) == B && XK.size(1) == H && XK.size(2) == N && XK.size(3) == F, "XK");
    TORCH_CHECK(XV.device().is_cuda() && XV.is_contiguous() && XV.dim() == 4 && XV.size(0) == B && XV.size(1) == H && XV.size(2) == N && XV.size(3) == F, "XV");
    TORCH_CHECK(W1.device().is_cuda() && W1.is_contiguous() && W1.dim() == 4 && W1.size(0) == B && W1.size(1) == H && W1.size(2) == F && W1.size(3) == F*K, "W1");
    TORCH_CHECK(W2.device().is_cuda() && W2.is_contiguous() && W2.dim() == 4 && W2.size(0) == B && W2.size(1) == H && W2.size(2) == F*K && W2.size(3) == F, "W2");
    TORCH_CHECK(out.device().is_cuda() && out.is_contiguous() && out.dim() == 4 && out.size(0) == B && out.size(1) == H && out.size(2) == N && out.size(3) == F, "out");

    bool *h_signal = (bool *)malloc(sizeof(bool)), *signal;
    hipMalloc(&signal, sizeof(bool));
    *h_signal = false;
    hipMemcpy(signal, h_signal, sizeof(bool), hipMemcpyHostToDevice);

    ttt_tp_forward_ker<B, H, N, F, K, TP><<<dim3(TP, B, H), NUM_WORKERS*kittens::WARP_THREADS>>>(
        gl<bf16, B, H, N, F>{reinterpret_cast<bf16*>(XQ.data_ptr<at::BFloat16>()), nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, N, F>{reinterpret_cast<bf16*>(XK.data_ptr<at::BFloat16>()), nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, N, F>{reinterpret_cast<bf16*>(XV.data_ptr<at::BFloat16>()), nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, F, F*K>{reinterpret_cast<bf16*>(W1.data_ptr<at::BFloat16>()), nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, F*K, F>{reinterpret_cast<bf16*>(W2.data_ptr<at::BFloat16>()), nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, N, F>{reinterpret_cast<bf16*>(out.data_ptr<at::BFloat16>()), nullptr, nullptr, nullptr, nullptr},
        signal
    );

    hipMemcpy(h_signal, signal, sizeof(bool), hipMemcpyDeviceToHost);
    TORCH_CHECK(*h_signal, "Kernel failed, *signal=true not set");

    return out;
}//*/

int main() {
    constexpr int B = 1, H = 1, N = 16, F = 64, K = 4, TP = 4;

    bf16 *h_XQ, *h_XK, *h_XV, *h_W1, *h_W2, *h_out;

    // Allocate host memory
    h_XQ = (bf16*)malloc(B*H*N*F*sizeof(bf16));
    h_XK = (bf16*)malloc(B*H*N*F*sizeof(bf16));
    h_XV = (bf16*)malloc(B*H*N*F*sizeof(bf16));
    h_W1 = (bf16*)malloc(B*H*F*F*K*sizeof(bf16));
    h_W2 = (bf16*)malloc(B*H*F*K*F*sizeof(bf16));
    h_out = (bf16*)malloc(B*H*N*F*sizeof(bf16));

    // Initialize host arrays
    for (int i = 0; i < B*H*N*F; i++) {
        h_XQ[i] = __int2bfloat16_rn(i);
        h_XK[i] = __int2bfloat16_rn(i);
        h_XV[i] = __int2bfloat16_rn(i);
        h_out[i] = __int2bfloat16_rn(-1);
    }
    for (int i = 0; i < B*H*F*F*K; i++) {
        h_W1[i] = __int2bfloat16_rn(i);
        h_W2[i] = __int2bfloat16_rn(i);
    }

    bf16 *XQ, *XK, *XV, *W1, *W2, *out;

    // Allocate device memory
    hipMalloc(&XQ, B*H*N*F*sizeof(bf16));
    hipMalloc(&XK, B*H*N*F*sizeof(bf16));
    hipMalloc(&XV, B*H*N*F*sizeof(bf16));
    hipMalloc(&W1, B*H*F*F*K*sizeof(bf16));
    hipMalloc(&W2, B*H*F*K*F*sizeof(bf16));
    hipMalloc(&out, B*H*N*F*sizeof(bf16));

    // Copy data from host to device
    hipMemcpy(XQ, h_XQ, B*H*N*F*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(XK, h_XK, B*H*N*F*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(XV, h_XV, B*H*N*F*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(W1, h_W1, B*H*F*F*K*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(W2, h_W2, B*H*F*K*F*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(out, h_out, B*H*N*F*sizeof(bf16), hipMemcpyHostToDevice);

    bool *h_signal = (bool *)malloc(sizeof(bool)), *signal;
    hipMalloc(&signal, sizeof(bool));
    *h_signal = false;
    hipMemcpy(signal, h_signal, sizeof(bool), hipMemcpyHostToDevice);

    printf("Launching kernel\n");

    ttt_tp_forward_ker<B, H, N, F, K, TP><<<dim3(TP, B, H), NUM_WORKERS*kittens::WARP_THREADS>>>(
        gl<bf16, B, H, N, F>{XQ, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, N, F>{XK, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, N, F>{XV, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, F, F*K>{W1, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, F*K, F>{W2, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, N, F>{out, nullptr, nullptr, nullptr, nullptr},
        signal
    );

    hipMemcpy(h_signal, signal, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT(*h_signal);

    printf("Ran successfully\n");
}