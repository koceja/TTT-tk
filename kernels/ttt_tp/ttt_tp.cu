#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include "hip/hip_cooperative_groups.h"
#include "kittens.cuh"

#define ASSERT(cond) if (!(cond)) { printf("Assertion failed: %s\n", #cond); return 1; }

using namespace kittens;

constexpr int NUM_WORKERS = 4; // TODO: why is one warpgroup optimal?

template<int B=1, int H=1, int N=16, int F=64, int K=4, int TP=4>
__launch_bounds__(NUM_WORKERS*WARP_THREADS, 1)
__cluster_dims__(TP)
__global__ void ttt_tp_ker(
    // TODO: make B and H runtime dimensions by instantiating templates with -1 args
    const __grid_constant__ gl<bf16, B, H, N, F> XQ_gl,
    const __grid_constant__ gl<bf16, B, H, N, F> XK_gl,
    const __grid_constant__ gl<bf16, B, H, N, F> XV_gl,
    const __grid_constant__ gl<bf16, B, H, F, F*K> W1_gl,
    const __grid_constant__ gl<bf16, B, H, F*K, F> W2_gl,
    bool *signal
) {
    int b = blockIdx.y;
    int h = blockIdx.z;
    cooperative_groups::cluster_group cluster = cooperative_groups::this_cluster();
    int tp = cluster.block_rank();
    
    rt_bf<N, F, ducks::rt_layout::row> XK;
    rt_bf<F, F*K/TP, ducks::rt_layout::col> W1;
    rt_fl<N, F*K/TP, ducks::rt_layout::row> Z1;
    rt_bf<N, F*K/TP, ducks::rt_layout::row> X2;
    rt_bf<F*K/TP, F, ducks::rt_layout::col> W2;
    rt_fl<N, F, ducks::rt_layout::row> out;
    st_fl<N, F> out_part;
    
    load(XK, XK_gl, {b, h, 0, 0});
    load(W1, W1_gl, {b, h, 0, tp*(F*K/TP)});
    zero(Z1);
    mma_AB(Z1, XK, W1, Z1);
    copy(X2, Z1); //dtype conversion
    mma_AB(out, X2, W2, out);
    
    // store(out_part, out);
    for (int i = 0; i < N*F; i+=blockDim.x)
        out_part[i + threadIdx.x] = tp;

    extern __shared__ KITTENS_DEFAULT_ALIGN bf16 shm[N * F];
    bf16 *dsmem = cluster.map_shared_rank(&shm[0], 0);
        for (int i = 0; i < N*F; i+=blockDim.x)
            atomicAdd(&dsmem[i + threadIdx.x], out_part[i + threadIdx.x]);
    cluster.sync();

    if (tp == 0 && threadIdx.x == 0)
        for (int i = 0; i < N*F; i++)
            printf("%.1f ", __bfloat162float(shm[i]));
    cluster.sync();

    *signal = true;
}

int main() {
    constexpr int B = 1, H = 1, N = 16, F = 64, K = 4, TP = 4;

    bf16 *h_XQ, *h_XK, *h_XV, *h_W1, *h_W2;

    // Allocate host memory
    h_XQ = (bf16*)malloc(B*H*N*F*sizeof(bf16));
    h_XK = (bf16*)malloc(B*H*N*F*sizeof(bf16));
    h_XV = (bf16*)malloc(B*H*N*F*sizeof(bf16));
    h_W1 = (bf16*)malloc(B*H*F*F*K*sizeof(bf16));
    h_W2 = (bf16*)malloc(B*H*F*K*F*sizeof(bf16));

    // Initialize host arrays
    for (int i = 0; i < B*H*N*F; i++) {
        h_XQ[i] = 0;
        h_XK[i] = i;
        h_XV[i] = i;
    }
    for (int i = 0; i < B*H*F*F*K; i++) {
        h_W1[i] = i;
        h_W2[i] = i;
    }

    bf16 *XQ, *XK, *XV, *W1, *W2;

    // Allocate device memory
    hipMalloc(&XQ, B*H*N*F*sizeof(bf16));
    hipMalloc(&XK, B*H*N*F*sizeof(bf16));
    hipMalloc(&XV, B*H*N*F*sizeof(bf16));
    hipMalloc(&W1, B*H*F*F*K*sizeof(bf16));
    hipMalloc(&W2, B*H*F*K*F*sizeof(bf16));

    // Copy data from host to device
    hipMemcpy(XQ, h_XQ, B*H*N*F*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(XK, h_XK, B*H*N*F*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(XV, h_XV, B*H*N*F*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(W1, h_W1, B*H*F*F*K*sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(W2, h_W2, B*H*F*K*F*sizeof(bf16), hipMemcpyHostToDevice);

    // Free host memory
    bool *h_signal = (bool *)malloc(sizeof(bool)), *signal;
    hipMalloc(&signal, sizeof(bool));
    *h_signal = false;
    hipMemcpy(signal, h_signal, sizeof(bool), hipMemcpyHostToDevice);

    printf("Launching kernel\n");

    ttt_tp_ker<B, H, N, F, K, TP><<<dim3(TP, B, H), NUM_WORKERS*kittens::WARP_THREADS>>>(
        gl<bf16, B, H, N, F>{XQ, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, N, F>{XK, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, N, F>{XV, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, F, F*K>{W1, nullptr, nullptr, nullptr, nullptr},
        gl<bf16, B, H, F*K, F>{W2, nullptr, nullptr, nullptr, nullptr},
        signal
    );

    hipMemcpy(h_signal, signal, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT(*h_signal);

    printf("Ran successfully\n");
}