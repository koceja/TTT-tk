#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "hip/hip_cooperative_groups.h"
#include <iostream>

// Build torch entrypoint
#ifdef TORCH_COMPILE
#define TK_COMPILE_TTT_MLP_FORWARD_TP
#endif

#define CUDA_ASSERT(cond, tidx) if (!(cond)) { if (tidx == -1 || threadIdx.x == tidx && tp_idx == 0) printf("Kernel assert failed: %s\n", #cond); return; }

constexpr int CONSUMER_WARPGROUPS = (1); 
constexpr int PRODUCER_WARPGROUPS = (1); 
constexpr int NUM_WARPGROUPS      = (CONSUMER_WARPGROUPS+PRODUCER_WARPGROUPS); 
constexpr int NUM_WORKERS         = (NUM_WARPGROUPS*kittens::WARPGROUP_WARPS); 
constexpr int TP                  = (4);

using namespace kittens;
namespace cg = cooperative_groups;

template<int D> struct fwd_attend_ker_tile_dims {};
template<> struct fwd_attend_ker_tile_dims<64> {
    constexpr static int tile_width = (64);
    constexpr static int tile_height = (64);
};

template<int D> struct fwd_globals {
    using q_tile    =         st_bf<fwd_attend_ker_tile_dims<D>::tile_height, fwd_attend_ker_tile_dims<D>::tile_width>;
    using k_tile    =         st_bf<fwd_attend_ker_tile_dims<D>::tile_height, fwd_attend_ker_tile_dims<D>::tile_width>;
    using v_tile    =         st_bf<fwd_attend_ker_tile_dims<D>::tile_height, fwd_attend_ker_tile_dims<D>::tile_width>;
    using o_tile    =         st_bf<fwd_attend_ker_tile_dims<D>::tile_height, fwd_attend_ker_tile_dims<D>::tile_width>;

    using w1_tile   =         st_bf<fwd_attend_ker_tile_dims<D>::tile_height, fwd_attend_ker_tile_dims<D>::tile_width>;
    using w2_tile   =         st_bf<fwd_attend_ker_tile_dims<D>::tile_height, fwd_attend_ker_tile_dims<D>::tile_width>;

    using q_gl = gl<bf16, -1, -1, -1, -1, q_tile>;
    using k_gl = gl<bf16, -1, -1, -1, -1, k_tile>;
    using v_gl = gl<bf16, -1, -1, -1, -1, v_tile>;
    using o_gl = gl<bf16, -1, -1, -1, -1, o_tile>;

    using w1_gl = gl<bf16, -1, -1, -1, -1, w1_tile>;
    using w2_gl = gl<bf16, -1, -1, -1, -1, w2_tile>;

    q_gl q;
    k_gl k;
    v_gl v;
    o_gl o;
    w1_gl w1;
    w2_gl w2;

    const int N;
};

template<int TP, ducks::st::all ST>
__device__ __forceinline__ void square_all_reduce(ST &tile, ST &tile_other, int tp) {
    if constexpr (TP == 1) {
        tma::cluster::arrive_aligned();
    } else {
        static_assert(TP == 4, "TP must be 4 for this square_all_reduce implementation");
        __shared__ semaphore dsmem_semaphore[2];

        if (warpgroup::warpid() == 0) {
            init_semaphore(dsmem_semaphore[0], 0, 1);
            tma::expect_bytes(dsmem_semaphore[0], sizeof(tile_other));
            init_semaphore(dsmem_semaphore[1], 0, 1);
            tma::expect_bytes(dsmem_semaphore[1], sizeof(tile_other));
        }
        tma::cluster::sync();

        for(int stage = 0; stage < 2; stage++) {
            if (warpgroup::warpid() == 0) {
                tma::cluster::store_async(tile_other, tile, tp ^ (1 << stage), dsmem_semaphore[stage]);
                kittens::wait(dsmem_semaphore[stage], 0);
            }
            warpgroup::sync(1);
            warpgroup::add(tile, tile, tile_other);
        }
    }
}

template<int D>
__global__  __launch_bounds__((NUM_WORKERS)*kittens::WARP_THREADS, 1)
__cluster_dims__(4)
void fwd_attend_ker(const __grid_constant__ fwd_globals<D> g) {
    extern __shared__ int __shm[]; 
    tma_swizzle_allocator al((int*)&__shm[0]);
    int warpid = kittens::warpid(), warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    using K = fwd_attend_ker_tile_dims<D>;

    using q_tile    =         st_bf<K::tile_height, K::tile_width>;
    using k_tile    =         st_bf<K::tile_height, K::tile_width>;
    using v_tile    =         st_bf<K::tile_height, K::tile_width>;

    using z1_tile   =         st_bf<K::tile_height, K::tile_width>;
    using z2_tile   =         st_bf<K::tile_height, K::tile_width>;
    using grad_z1_tile   =         st_bf<K::tile_height, K::tile_width>;
    using rd_buffer_tile   =         st_bf<K::tile_height, K::tile_width>;

    using w1_tile   =         st_bf<K::tile_height, K::tile_width>;
    using w2_tile   =         st_bf<K::tile_height, K::tile_width>;

    w1_tile    (&w1_smem)                   = al.allocate<w1_tile>();
    w2_tile    (&w2_smem)                   = al.allocate<w2_tile>();

    q_tile    (&q_smem)                    = al.allocate<q_tile>();
    k_tile    (&k_smem)                    = al.allocate<k_tile>();
    v_tile    (&v_smem)                    = al.allocate<v_tile>();

    z1_tile    (&z1_smem)                   = al.allocate<z1_tile>();
    z2_tile    (&z2_smem)                   = al.allocate<z2_tile>();
    grad_z1_tile    (&grad_z1_smem)         = al.allocate<grad_z1_tile>();
    rd_buffer_tile    (&rd_buffer_smem)      = al.allocate<rd_buffer_tile>();

    int batch_idx   = blockIdx.y;
    int head_idx    = blockIdx.z;
    int n_minibatch   = g.N / (K::tile_height);

    cooperative_groups::cluster_group cluster = cooperative_groups::this_cluster();
    int tp_idx = cluster.block_rank();
    CUDA_ASSERT(tp_idx == blockIdx.x, 0);

    __shared__ kittens::semaphore w1_smem_arrived, w2_smem_arrived, q_sem_arrived, k_sem_arrived, v_sem_arrived, reduction_done, compute_done;
    if (threadIdx.x == 0) { 
        init_semaphore(w1_smem_arrived, 0, 1);
        init_semaphore(w2_smem_arrived, 0, 1);
        init_semaphore(q_sem_arrived, 0, 1); 
        init_semaphore(k_sem_arrived, 0, 1); 
        init_semaphore(v_sem_arrived, 0, 1); 
        init_semaphore(reduction_done, CONSUMER_WARPGROUPS, 0);
        init_semaphore(compute_done, CONSUMER_WARPGROUPS, 0);

        tma::expect_bytes(w1_smem_arrived, sizeof(w1_tile));
        tma::load_async(w1_smem, g.w1, {batch_idx, head_idx, 0, tp_idx}, w1_smem_arrived);

        tma::expect_bytes(w2_smem_arrived, sizeof(w2_tile));
        tma::load_async(w2_smem, g.w2, {batch_idx, head_idx, tp_idx, 0}, w2_smem_arrived);

        int4 tile_idx = {batch_idx, head_idx, 0, 0};

        tma::expect_bytes(q_sem_arrived, sizeof(q_tile));
        tma::load_async(q_smem, g.q, tile_idx, q_sem_arrived);

        tma::expect_bytes(k_sem_arrived, sizeof(k_tile));
        tma::load_async(k_smem, g.k, tile_idx, k_sem_arrived);

        tma::expect_bytes(v_sem_arrived, sizeof(v_tile));
        tma::load_async(v_smem, g.v, tile_idx, v_sem_arrived);
    }
    __syncthreads(); 

    if(warpgroupid == NUM_WARPGROUPS-1) {
        warpgroup::decrease_registers<32>();    
        tma::cluster::arrive_aligned();
        
        int iters; 
        iters = n_minibatch - 1;

        kittens::wait(reduction_done, 0);
        if (warpid == NUM_WORKERS-4) {
            for (auto idx = 0; idx < iters; idx++) {
                kittens::wait(reduction_done, idx % 2);
                kittens::wait(compute_done, idx % 2);
                
                int4 tile_idx = {batch_idx, head_idx, idx + 1, 0};

                tma::expect_bytes(q_sem_arrived, sizeof(q_tile));
                tma::load_async(q_smem, g.q, tile_idx, q_sem_arrived);

                tma::expect_bytes(k_sem_arrived, sizeof(k_tile));
                tma::load_async(k_smem, g.k, tile_idx, k_sem_arrived);

                tma::expect_bytes(v_sem_arrived, sizeof(v_tile));
                tma::load_async(v_smem, g.v, tile_idx, v_sem_arrived);

                // warpgroup::sync(NUM_WARPGROUPS-1); TODO: Is this needed?
                tma::cluster::arrive_aligned();
            }
        }
    }
    else {
        warpgroup::increase_registers<184>();

        rt_fl<16, K::tile_height> cs_cs_fl_reg;
        rt_fl<16, K::tile_height> cs_cs_2_fl_reg;
        rt_bf<16, K::tile_height> cs_cs_bf_reg;

        kittens::wait(w1_smem_arrived, 0);
        kittens::wait(w2_smem_arrived, 0);
        
        for (auto idx = 0; idx < n_minibatch; idx++) {
            // Hidden State Forward
            kittens::wait(k_sem_arrived, idx % 2);

            warpgroup::mm_AB(cs_cs_fl_reg, k_smem, w1_smem);
            warpgroup::mma_async_wait();
            warpgroup::store(z1_smem, cs_cs_fl_reg);

            warpgroup::mm_AB(cs_cs_fl_reg, z1_smem, w2_smem);
            warpgroup::mma_async_wait();
            warpgroup::store(z2_smem, cs_cs_fl_reg);

            // Reduction over SM
            square_all_reduce<TP>(z2_smem, rd_buffer_smem, tp_idx);
            if (warpgroup::laneid() == 0) arrive(reduction_done, 1);

            // Calculate (negative) grad_l_wrt_Z2 / grad_l_wrt_Z1
            // We use negative gradients to use the WGMMA accumulator
            kittens::wait(v_sem_arrived, idx % 2);
            warpgroup::sub(z2_smem, z2_smem, v_smem); // grad_l_wrt_Z2 is stored into z2_smem
            warpgroup::mm_ABt(cs_cs_fl_reg, z2_smem, w2_smem);
            warpgroup::mma_async_wait();
            warpgroup::store(grad_z1_smem, cs_cs_fl_reg);

            // Compute Attn1 and Z1_bar partial (on registers)
            kittens::wait(q_sem_arrived, idx % 2);
            warpgroup::mm_ABt(cs_cs_fl_reg, q_smem, k_smem);
            warpgroup::mm_AB(cs_cs_2_fl_reg, q_smem, w1_smem); // Z1_bar partial
            
            // Compute Z1_bar using Z1_bar partial (on registers)
            copy(cs_cs_bf_reg, cs_cs_fl_reg);
            make_causal(cs_cs_bf_reg, cs_cs_bf_reg, base_types::constants<bf16>::zero());
            warpgroup::mma_AB(cs_cs_2_fl_reg, cs_cs_bf_reg, grad_z1_smem); // Z1_bar
            warpgroup::mma_async_wait();

            // Compute Attn2 and Z2_bar partial (on registers)
            copy(cs_cs_bf_reg, cs_cs_2_fl_reg);
            warpgroup::mm_ABt(cs_cs_fl_reg, cs_cs_bf_reg, z1_smem); // Attn2
            warpgroup::mm_AB(cs_cs_2_fl_reg, cs_cs_bf_reg, w2_smem); // Z2_bar partial
            warpgroup::mma_async_wait();

            // Compute Z2_bar using Z2_bar partial (on registers)
            copy(cs_cs_bf_reg, cs_cs_fl_reg);
            make_causal(cs_cs_bf_reg, cs_cs_bf_reg, base_types::constants<bf16>::zero());
            warpgroup::mma_AB(cs_cs_2_fl_reg, cs_cs_bf_reg, z2_smem); // Z2_bar
            warpgroup::mma_async_wait();

            // Store Z2_bar into global memory
            warpgroup::store(z2_smem, cs_cs_2_fl_reg);
            if (warpgroup::warpid() == 0) {
                tma::store_add_async(g.o, z2_smem, {batch_idx, head_idx, idx, 0});
                tma::store_commit_group();
            }

            // Update hidden states
            warpgroup::load(cs_cs_fl_reg, w1_smem);
            warpgroup::mma_AtB(cs_cs_fl_reg, k_smem, grad_z1_smem);
            warpgroup::mma_async_wait();
            warpgroup::store(w1_smem, cs_cs_fl_reg);

            warpgroup::load(cs_cs_fl_reg, w2_smem);
            warpgroup::mma_AtB(cs_cs_fl_reg, z1_smem, z2_smem);
            warpgroup::mma_async_wait();
            warpgroup::store(w2_smem, cs_cs_fl_reg);

            if (warpgroup::laneid() == 0) arrive(compute_done, 1);
        }
    }
}

// Modified ttt_mlp_forward function
#ifdef TK_COMPILE_TTT_MLP_FORWARD_TP
#include "common/pyutils/torch_helpers.cuh"
void ttt_mlp_forward_tp(
    // const torch::Tensor ttt_norm_weight,
    // const torch::Tensor ttt_norm_bias,
    const torch::Tensor W1_init,
    // const torch::Tensor b1_init,
    const torch::Tensor W2_init,
    // const torch::Tensor b2_init,
    const torch::Tensor XQ_batch,
    const torch::Tensor XV_batch,
    const torch::Tensor XK_batch,
    torch::Tensor output
    // const torch::Tensor eta_batch
)
{
    // Initalize data pointers
    auto *d_q = reinterpret_cast<bf16*>(XQ_batch.data_ptr<at::BFloat16>());
    auto *d_k = reinterpret_cast<bf16*>(XV_batch.data_ptr<at::BFloat16>());
    auto *d_v = reinterpret_cast<bf16*>(XK_batch.data_ptr<at::BFloat16>());
    auto *d_w1 = reinterpret_cast<bf16*>(W1_init.data_ptr<at::BFloat16>());
    auto *d_w2 = reinterpret_cast<bf16*>(W2_init.data_ptr<at::BFloat16>());
    auto *d_o = reinterpret_cast<bf16*>(output.data_ptr<at::BFloat16>());

    constexpr int BATCH_SIZE = 1;
    constexpr int HEADS = 1;
    constexpr int TP = 4;

    constexpr int SEQ_LEN = 64; 
    constexpr int HEAD_DIM = 64; 
    constexpr int EXP_DIM = 256;
    constexpr int BLOCK_SIZE = (NUM_WORKERS*32); // Number of threads in a block

    using globals = fwd_globals<HEAD_DIM>;

    globals::q_gl qg_arg{d_q, BATCH_SIZE, HEADS, SEQ_LEN, HEAD_DIM};
    globals::k_gl kg_arg{d_k, BATCH_SIZE, HEADS, SEQ_LEN, HEAD_DIM};
    globals::v_gl vg_arg{d_v, BATCH_SIZE, HEADS, SEQ_LEN, HEAD_DIM};
    globals::o_gl og_arg{d_o, BATCH_SIZE, HEADS, SEQ_LEN, HEAD_DIM};

    globals::w1_gl w1g_arg{d_w1, BATCH_SIZE, HEADS, HEAD_DIM, EXP_DIM};
    globals::w2_gl w2g_arg{d_w2, BATCH_SIZE, HEADS, EXP_DIM, HEAD_DIM};

    globals g{qg_arg, kg_arg, vg_arg, w1g_arg, w2g_arg, og_arg, SEQ_LEN};
    
    // Set shared memory to use max dynamic
    unsigned long mem_size = kittens::MAX_SHARED_MEMORY; // need to launch two blocks if possible.
    
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        fwd_attend_ker<HEAD_DIM>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );

    dim3 grid(TP, BATCH_SIZE, HEADS);

    hipDeviceSynchronize();
    fwd_attend_ker<HEAD_DIM><<<grid, BLOCK_SIZE, mem_size>>>(g);
    hipDeviceSynchronize();
}
#endif

#include "harness.impl"
